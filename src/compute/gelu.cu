#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

#include "compute/gelu.h"
#include "util.h"

namespace dllm::compute {
namespace {
template <typename T>
__global__ void GeLU(T* __restrict__ output, const T* __restrict__ input,
                     std::size_t n) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= n) {
    return;
  }

  // For double use double erf, for single, half, bfloat16 use single efr
  constexpr auto useDouble = sizeof(T) > sizeof(float);
  using TargetType = std::conditional_t<useDouble, double, float>;
  TargetType inputElement = input[tid];

  constexpr auto inv_sqrt_2 = 0.7071067811865475;
  output[tid] = static_cast<TargetType>(0.5) * inputElement *
                (static_cast<TargetType>(1.) +
                 erf(inputElement * static_cast<TargetType>(HIP_SQRT_HALF)));
}

template <typename Fn>
__inline__ __attribute__((always_inline)) void autoDispatch(Dtype dtype,
                                                            Fn&& fn) {
  switch (dtype) {
    case R_64F:
      fn(double{0});
      return;
    case R_32F:
      fn(float{0});
      return;
    case R_16F:
      fn(nv_half{0});
      return;
    case R_16BF:
      fn(hip_bfloat16{0});
      return;
    default:
      return;
  }
}
}  // namespace

void GeLUKernel(hipStream_t cudaStream, Tensor1D& output,
                const Tensor1D& input) {
  const auto size = cute::size(input.layout);
  auto f = [&](auto dummy) {
    using T = std::remove_const_t<std::decay_t<decltype(dummy)>>;
    dim3 block(std::min(128, size));
    dim3 grid(util::ceilDiv(size, std::min(128, size)));
    GeLU<<<grid, block, 0, cudaStream>>>(static_cast<T*>(output.data()),
                                         static_cast<const T*>(input.data()),
                                         size);
  };
  autoDispatch(output.dtype, f);
}
}  // namespace dllm::compute
