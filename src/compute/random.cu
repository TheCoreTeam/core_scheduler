#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "compute/random.h"
#include "random/random_internal.h"
#include "util.h"

namespace dllm::compute::Random {
namespace {
template <typename T>
__global__ void gaussian(T *y, const unsigned long curandSeed,
                         const unsigned long curandOffset, std::size_t n) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= n) {
    return;
  }
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(curandSeed, tid, curandOffset, &state);
  y[tid] = static_cast<T>(hiprand_normal(&state));
}

template <typename T>
__global__ void uniform(T *y, const unsigned long curandSeed,
                        const unsigned long curandOffset, std::size_t n) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= n) {
    return;
  }
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(curandSeed, tid, curandOffset, &state);
  y[tid] = static_cast<T>(hiprand_uniform(&state));
}

template <typename Fn>
__inline__ __attribute__((always_inline)) void autoDispatch(Dtype dtype,
                                                            Fn &&fn) {
  switch (dtype) {
    case R_64F:
      fn(double{0});
      return;
    case R_32F:
      fn(float{0});
      return;
    case R_16F:
      fn(nv_half{0});
      return;
    case R_16BF:
      fn(hip_bfloat16{0});
      return;
    default:
      return;
  }
}
}  // namespace

void gaussianKernel(const ContextCompute *context, Tensor1D &tensor) {
  const auto size = cute::size(tensor.layout);
  auto &[seed, offset] = random::getRandomState();
  auto f = [&](auto dummy) {
    using T = std::remove_const_t<std::decay_t<decltype(dummy)>>;
    dim3 block(std::min<decltype(size)>(128, size));
    dim3 grid(util::ceilDiv(size, std::min<decltype(size)>(128, size)));
    gaussian<<<grid, block, 0, context->cudaStream>>>(
        static_cast<T *>(tensor.data()), seed, offset.fetch_add(size), size);
  };
  autoDispatch(tensor.dtype, f);
}

void uniformKernel(const ContextCompute *context, Tensor1D &tensor) {
  const auto size = cute::size(tensor.layout);
  auto &[seed, offset] = random::getRandomState();
  auto f = [&](auto dummy) {
    using T = std::remove_const_t<std::decay_t<decltype(dummy)>>;
    dim3 block(std::min<decltype(size)>(128, size));
    dim3 grid(util::ceilDiv(size, std::min<decltype(size)>(128, size)));
    uniform<<<grid, block, 0, context->cudaStream>>>(
        static_cast<T *>(tensor.data()), seed, offset.fetch_add(size), size);
  };
  autoDispatch(tensor.dtype, f);
}
}  // namespace dllm::compute::Random
